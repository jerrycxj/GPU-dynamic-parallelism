#include <algorithm> // std::swap/
#include <assert.h>
#include <iostream>
#include <omp.h>
#include <png.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define GRID_CODE 999
#define SAVE_FRACTAL 0
#define SAVE_GRIDLINES 1

#include "stats.cuh"
#include "gridlines.cuh"
#include "ask.cuh"
#include "askNEW.cuh"
#include "exhaustive.cuh"
#include "complex.cuh"
#include "dynamicParallelism.cuh"
#include "macros.cuh"
#include "mandelbrotHelper.cuh"
#include "tools.cuh"
#include "doTests.cuh"


using namespace std;
const char* approachStr[4] = {"Ex", "DP", "ASK-v1", "ASK-v2"};

/** gets the color, given the dwell */
void dwell_color(int *r, int *g, int *b, int dwell, unsigned int CA_MAXDWELL);

int main(int argc, char **argv) {

    check_args_info(argc);

    char approach = atoi(argv[1]);
    unsigned int W = stoi(argv[2]);
    unsigned int H = stoi(argv[3]);
    float rmin = atof(argv[4]);
    float rmax = atof(argv[5]);
    float cmin = atof(argv[6]);
    float cmax = atof(argv[7]);

    int CA_MAXDWELL = atoi(argv[8]);
    int B = atoi(argv[9]);
    int g0 = atoi(argv[10]);
    int r = atoi(argv[11]);
    int MAX_DEPTH = atoi(argv[12]);
    string fileName = argv[13];

    int *h_dwells;
    int *d_dwells;


    complex bottomLeftCorner = complex(rmin, cmin);
    complex upperRightCorner = complex(rmax, cmax);
    size_t dwell_sz = (size_t)W * H * sizeof(int);
    #ifdef VERBOSE
        float domainGBytes = (float)(sizeof(unsigned int) * W * H)/(1024*1024*1024);
        printf("\nGrid..............................................%i x %i (%.2f GiB)\n", W, H, domainGBytes);
    #endif


    // ---------------------
    // 1) memory allocation
    // ---------------------
    cucheck(hipMalloc((void **)&d_dwells, dwell_sz));
    hipDeviceSynchronize();
    h_dwells = (int *)malloc(dwell_sz);




    // ---------------------
    // 2) GPU Compute
    // ---------------------
    #ifdef VERBOSE
        printf("%s (REPEATS=%3i, REALIZATIONS=%3i)............", approachStr[approach], REPEATS, REALIZATIONS); fflush(stdout);
    #endif
    statistics stat = doTest(approach, d_dwells, W, H, bottomLeftCorner, upperRightCorner, g0, r, CA_MAXDWELL, B, MAX_DEPTH);
    hipDeviceSynchronize();
    #ifdef VERBOSE
        printf("done: %f secs (stErr %f%%)\n", stat.mean, 100.0*stat.sterr/stat.mean); fflush(stdout);
    #endif




    // ----------------------------
    // 3) copy domain back to Host
    // ----------------------------
    #ifdef VERBOSE
        printf("hipMemcpy: Host <-- Dev (%5.2f GiB)..............", domainGBytes); fflush(stdout);
    #endif
    cucheck(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
    #ifdef VERBOSE
        printf("done\n"); fflush(stdout);
    #endif



    // -------------------
    // 4) Export Fractal Image
    // -------------------
    if (fileName != "none"){
        string fractalFileName = fileName + string(".png");
        #ifdef VERBOSE
            printf("Saving %s.................................", fractalFileName.c_str()); fflush(stdout);
        #endif
        save_image(fractalFileName.c_str(), h_dwells, W, H, CA_MAXDWELL, SAVE_FRACTAL);
        #ifdef VERBOSE
            printf("done\n"); fflush(stdout);
        #endif
    }


    // -----------------------
    // Export gridlines image (does computation)
    // -----------------------
    #ifdef GRIDLINES
        // ------------------
        // compute gridlines
        // ------------------
        #ifdef VERBOSE
            printf("GridLines........................................."); fflush(stdout);
        #endif
        float gridTime = doGridLines( d_dwells, W, H, bottomLeftCorner, upperRightCorner, g0, r, CA_MAXDWELL, B, MAX_DEPTH);
        hipDeviceSynchronize();
        #ifdef VERBOSE
            printf("done: %f secs\n", gridTime); fflush(stdout);
        #endif



        // ----------------------------
        // copy gridlines back to host
        // ----------------------------
        #ifdef VERBOSE
            printf("hipMemcpy: Host <-- Dev (%5.2f GiB)..............", domainGBytes); fflush(stdout);
        #endif
        cucheck(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
        #ifdef VERBOSE
            printf("done\n"); fflush(stdout);
        #endif



        // ---------------------
        // save gridlines image
        // ---------------------
        if (fileName != "none"){
            string gridFileName = fileName + string("-gridlines.png");
            #ifdef VERBOSE
                printf("Saving %s......", gridFileName.c_str()); fflush(stdout);
            #endif
            save_image( gridFileName.c_str(), h_dwells, W, H, CA_MAXDWELL, SAVE_GRIDLINES);
            #ifdef VERBOSE
                printf("done\n"); fflush(stdout);
            #endif
        }
    #endif
    #ifdef VERBOSE
        printf("\n");
    #endif

    //printf("%i,%s,   %i, %i,   %i, %i,   %i, %i,   %i, %i, %i,   %f, %f, %f, %f\n", 
    //        approach, approachStr[approach], BSX, BSY, W, H, CA_MAXDWELL, MAX_DEPTH, g0, r, B, 
    //        stat.mean, stat.stdev, stat.sterr, 100.0*stat.sterr/stat.mean);
    printf("%i,%f,%f,%f,%f", approach, stat.mean, stat.stdev, stat.sterr, 100.0*stat.sterr/stat.mean);
    exit(EXIT_SUCCESS);
}
