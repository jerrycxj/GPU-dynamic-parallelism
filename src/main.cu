#include <algorithm> // std::swap/
#include <assert.h>
#include <iostream>
#include <omp.h>
#include <png.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define GRID_CODE 999
#define SAVE_FRACTAL 0
#define SAVE_GRIDLINES 1

#include "stats.cuh"
#include "complex.cuh"
#include "gridlines.cuh"
#include "macros.cuh"
#include "mandelbrotHelper.cuh"
#include "tools.cuh"

#include "exhaustive.cuh"
#include "dp_mbr.cuh"
#include "dp_sbr.cuh"
#include "ask_sbr.cuh"
#include "ask_mbr.cuh"
#include "benchmark.cuh"



using namespace std;
const char* approachStr[5] = {"Ex", "DP-SBR", "DP-MBR", "ASK-SBR", "ASK-MBR"};

/** gets the color, given the dwell */
void dwell_color(int *r, int *g, int *b, int dwell, unsigned int CA_MAXDWELL);

int main(int argc, char **argv) {

    check_args_info(argc);

    char approach = atoi(argv[1]);
    unsigned int W = stoi(argv[2]);
    unsigned int H = stoi(argv[3]);
    float rmin = atof(argv[4]);
    float rmax = atof(argv[5]);
    float cmin = atof(argv[6]);
    float cmax = atof(argv[7]);

    int CA_MAXDWELL = atoi(argv[8]);
    int g = atoi(argv[9]);
    int r = atoi(argv[10]);
    int B = atoi(argv[11]);
    int MAX_DEPTH = atoi(argv[12]);
    string fileName = argv[13];

    int *h_dwells;
    int *d_dwells;


    complex bottomLeftCorner = complex(rmin, cmin);
    complex upperRightCorner = complex(rmax, cmax);
    size_t dwell_sz = (size_t)W * H * sizeof(int);
    #ifdef VERBOSE
        float domainGBytes = (float)(sizeof(unsigned int) * W * H)/(1024*1024*1024);
        printf("\nGrid..............................................%i x %i (%.2f GiB)\n", W, H, domainGBytes);
        printf("g=%i r=%i B=%i\n", g, r, B);
    #endif


    // ---------------------
    // 1) memory allocation
    // ---------------------
    cucheck(hipMalloc((void **)&d_dwells, dwell_sz));
    hipDeviceSynchronize();
    h_dwells = (int *)malloc(dwell_sz);




    // ---------------------
    // 2) GPU Compute
    // ---------------------
    #ifdef VERBOSE
        printf("%s (REALIZATIONS=%3i, REPEATS=%3i)............", approachStr[approach], REALIZATIONS, REPEATS); fflush(stdout);
    #endif
    statistics stat = doBenchmark(approach, d_dwells, W, H, bottomLeftCorner, upperRightCorner, g, r, CA_MAXDWELL, B, MAX_DEPTH);
    hipDeviceSynchronize();
    #ifdef VERBOSE
        printf("done: %f secs (stErr %f%%)\n", stat.mean, 100.0*stat.sterr/stat.mean); fflush(stdout);
    #endif




    // ----------------------------
    // 3) copy domain back to Host
    // ----------------------------
    #ifdef VERBOSE
        printf("hipMemcpy: Host <-- Dev (%5.2f GiB)..............", domainGBytes); fflush(stdout);
    #endif
    cucheck(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
    #ifdef VERBOSE
        printf("done\n"); fflush(stdout);
    #endif



    // -------------------
    // 4) Export Fractal Image
    // -------------------
    if (fileName != "none"){
        string fractalFileName = fileName + string(".png");
        #ifdef VERBOSE
            printf("Saving %s.................................", fractalFileName.c_str()); fflush(stdout);
        #endif
        save_image(fractalFileName.c_str(), h_dwells, W, H, CA_MAXDWELL, SAVE_FRACTAL);
        #ifdef VERBOSE
            printf("done\n"); fflush(stdout);
        #endif
    }


    // -----------------------
    // Export gridlines image (does computation)
    // -----------------------
    #ifdef GRIDLINES
        // ------------------
        // compute gridlines
        // ------------------
        #ifdef VERBOSE
            printf("GridLines........................................."); fflush(stdout);
        #endif
        float gridTime = doGridLines( d_dwells, W, H, bottomLeftCorner, upperRightCorner, g, r, CA_MAXDWELL, B, MAX_DEPTH);
        hipDeviceSynchronize();
        #ifdef VERBOSE
            printf("done: %f secs\n", gridTime); fflush(stdout);
        #endif



        // ----------------------------
        // copy gridlines back to host
        // ----------------------------
        #ifdef VERBOSE
            printf("hipMemcpy: Host <-- Dev (%5.2f GiB)..............", domainGBytes); fflush(stdout);
        #endif
        cucheck(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
        #ifdef VERBOSE
            printf("done\n"); fflush(stdout);
        #endif



        // ---------------------
        // save gridlines image
        // ---------------------
        if (fileName != "none"){
            string gridFileName = fileName + string("-gridlines.png");
            #ifdef VERBOSE
                printf("Saving %s......", gridFileName.c_str()); fflush(stdout);
            #endif
            save_image( gridFileName.c_str(), h_dwells, W, H, CA_MAXDWELL, SAVE_GRIDLINES);
            #ifdef VERBOSE
                printf("done\n"); fflush(stdout);
            #endif
        }
    #endif
    #ifdef VERBOSE
        printf("\n");
    #endif

    //printf("%i,%s,   %i, %i,   %i, %i,   %i, %i,   %i, %i, %i,   %f, %f, %f, %f\n", 
    //        approach, approachStr[approach], BSX, BSY, W, H, CA_MAXDWELL, MAX_DEPTH, g, r, B, 
    //        stat.mean, stat.stdev, stat.sterr, 100.0*stat.sterr/stat.mean);
    printf("%i,%f,%f,%f,%f", approach, stat.mean, stat.stdev, stat.sterr, 100.0*stat.sterr/stat.mean);
    exit(EXIT_SUCCESS);
}
