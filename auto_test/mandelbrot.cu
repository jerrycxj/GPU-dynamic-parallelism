
#include <hip/hip_runtime.h>
#include <assert.h>
#include <png.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <algorithm>    // std::swap


/** CUDA check macro */
#define cucheck(call) \
	{\
	hipError_t res = (call);\
	if(res != hipSuccess) {\
	const char* err_str = hipGetErrorString(res);\
	fprintf(stderr, "%s (%d): %s in %s", __FILE__, __LINE__, err_str, #call);	\
	exit(-1);\
	}\
	}

#define cucheck_dev(call) \
	{\
	hipError_t res = (call);\
	if(res != hipSuccess) {\
	const char* err_str = hipGetErrorString(res);\
	printf("%s (%d): %s in %s", __FILE__, __LINE__, err_str, #call);	\
	assert(0);																												\
	}\
	}


/** a useful function to compute the number of threads */
__host__ __device__ int divup(int x, int y) { return x / y + (x % y ? 1 : 0); }

/** gets the color, given the dwell */
void dwell_color(int *r, int *g, int *b, int dwell);

/** save the dwell into a PNG file 
		@remarks: code to save PNG file taken from here 
		  (error handling is removed):
		http://www.labbookpages.co.uk/software/imgProc/libPNG.html
 */
void save_image(const char *filename, int *dwells, int w, int h) {
	png_bytep row;
	
	FILE *fp = fopen(filename, "wb");
	png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, 0, 0, 0);
	png_infop info_ptr = png_create_info_struct(png_ptr);
	// exception handling
	setjmp(png_jmpbuf(png_ptr));
	png_init_io(png_ptr, fp);
	// write header (8 bit colour depth)
	png_set_IHDR(png_ptr, info_ptr, w, h,
							 8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
							 PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);
	// set title
	png_text title_text;
	title_text.compression = PNG_TEXT_COMPRESSION_NONE;
	title_text.key = "Title";
	title_text.text = "Mandelbrot set, per-pixel";
	png_set_text(png_ptr, info_ptr, &title_text, 1);
	png_write_info(png_ptr, info_ptr);

	// write image data
	row = (png_bytep) malloc(3 * w * sizeof(png_byte));
	for (int y = 0; y < h; y++) {
		for (int x = 0; x < w; x++) {
			int r, g, b;
			dwell_color(&r, &g, &b, dwells[y * w + x]);
			row[3 * x + 0] = (png_byte)r;
			row[3 * x + 1] = (png_byte)g;
			row[3 * x + 2] = (png_byte)b;
		}
		png_write_row(png_ptr, row);
	}
	png_write_end(png_ptr, NULL);

  fclose(fp);
  png_free_data(png_ptr, info_ptr, PNG_FREE_ALL, -1);
  png_destroy_write_struct(&png_ptr, (png_infopp)NULL);
  free(row);
}  // save_image


/** a simple complex type */
struct complex {
	__host__ __device__ complex(float re, float im = 0) {
		this->re = re;
		this->im = im;
	}
	/** real and imaginary part */
	float re, im;
}; // struct complex

// operator overloads for complex numbers
inline __host__ __device__ complex operator+
(const complex &a, const complex &b) {
	return complex(a.re + b.re, a.im + b.im);
}
inline __host__ __device__ complex operator-
(const complex &a) { return complex(-a.re, -a.im); }
inline __host__ __device__ complex operator-
(const complex &a, const complex &b) {
	return complex(a.re - b.re, a.im - b.im);
}
inline __host__ __device__ complex operator*
(const complex &a, const complex &b) {
	return complex(a.re * b.re - a.im * b.im, a.im * b.re + a.re * b.im);
}
inline __host__ __device__ float abs2(const complex &a) {
	return a.re * a.re + a.im * a.im;
}
inline __host__ __device__ complex operator/
(const complex &a, const complex &b) {
	float invabs2 = 1 / abs2(b);
	return complex((a.re * b.re + a.im * b.im) * invabs2,
								 (a.im * b.re - b.im * a.re) * invabs2);
}  // operator/

#ifndef MAX_DWELL
#define MAX_DWELL 512
#endif
/** block size along */
#ifndef BSX
#define BSX 16
#endif

#ifndef BSY
#define BSY 16
#endif
/** maximum recursion depth */
#ifndef MAX_DEPTH
#define MAX_DEPTH 5
#endif
/** region below which do per-pixel */
#define MIN_SIZE 4
/** subdivision factor along each axis */

#ifndef SUBDIV
#define SUBDIV 2
#endif

#ifndef SUBDIV_ELEMSX
#define SUBDIV_ELEMSX 1 //Potencia de 2 mas cercana a subdiv, restado en 1.
#endif

#ifndef SUBDIV_ELEMSP 
#define SUBDIV_ELEMSP 2 // Exponente de la potencia de 2 inmediatamente mayor que subdiv
#endif

#ifndef SUBDIV_ELEMS
#define SUBDIV_ELEMS 64//SUBDIV^2
#endif

#ifndef SUBDIV_ELEMS2
#define SUBDIV_ELEMS2 128 //SUBDIV^2
#endif
/** subdivision when launched from host */
#define INIT_SUBDIV 8

/** find the dwell for the pixel */
__device__ int pixel_dwell(int w, int h, complex cmin, complex cmax, int x, int y) {
	complex dc = cmax - cmin;
	float fx = (float)x / w, fy = (float)y / h;
	complex c = cmin + complex(fx * dc.re, fy * dc.im);
	int dwell = 0;
	complex z = c;
	while(dwell < MAX_DWELL && abs2(z) < 2 * 2) {
		z = z * z + c;
		dwell++;
	}
	return dwell;
}  // pixel_dwell

/** binary operation for common dwell "reduction": MAX_DWELL + 1 = neutral
		element, -1 = dwells are different */
#define NEUT_DWELL (MAX_DWELL + 1)
#define DIFF_DWELL (-1)
__device__ int same_dwell(int d1, int d2) {
	if(d1 == d2)
		return d1;
	else if(d1 == NEUT_DWELL || d2 == NEUT_DWELL)
		return min(d1, d2);
	else
		return DIFF_DWELL;
}  // same_dwell

/** evaluates the common border dwell, if it exists */
__device__ int border_dwell
(int* dwells, int w, int h, complex cmin, complex cmax, int x0, int y0, int d) {
	// check whether all boundary pixels have the same dwell
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int bs = blockDim.x * blockDim.y;
	int comm_dwell = NEUT_DWELL;
	// for all boundary pixels, distributed across threads
	for(int r = tid; r < d; r += bs) {
		// for each boundary: b = 0 is east, then counter-clockwise
		for(int b = 0; b < 4; b++) {
			int x = b % 2 != 0 ? x0 + r : (b == 0 ? x0 + d - 1 : x0);
			int y = b % 2 == 0 ? y0 + r : (b == 1 ? y0 + d - 1 : y0);
			int dwell = pixel_dwell(w, h, cmin, cmax, x, y);
			comm_dwell = same_dwell(comm_dwell, dwell);
            //dwells[y * w + x] = 666;//pixel_dwell(w, h, cmin, cmax, x, y);
		}
	}  // for all boundary pixels
	// reduce across threads in the block
	__shared__ int ldwells[BSX * BSY];
	int nt = min(d, BSX * BSY);
	if(tid < nt)
		ldwells[tid] = comm_dwell;
	__syncthreads();
	for(; nt > 1; nt /= 2) {
		if(tid < nt / 2)
			ldwells[tid] = same_dwell(ldwells[tid], ldwells[tid + nt / 2]);
		__syncthreads();
	}
	return ldwells[0];
}  // border_dwell

/** the kernel to fill the image region with a specific dwell value */
__global__ void dwell_fill_k
(int *dwells, int w, int x0, int y0, int d, int dwell) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < d && y < d) {
		x += x0, y += y0; 
        //if (dwells[y * w + x] != 666)
		dwells[y * w + x] = dwell;
	}
}  // dwell_fill_k

/** the kernel to fill in per-pixel values of the portion of the Mandelbrot set
		*/
__global__ void mandelbrot_pixel_k
(int *dwells, int w, int h, complex cmin, complex cmax, int x0, int y0, int d) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	if(x < d && y < d) {
		x += x0, y += y0;
        //if (dwells[y * w + x] != 666)
		dwells[y * w + x] = pixel_dwell(w, h, cmin, cmax, x, y);
	}
}  // mandelbrot_pixel_k

/** checking for an error */
__device__ void check_error(int x0, int y0, int d) {
	int err = hipGetLastError();
	if(err != hipSuccess) {
		printf("error launching kernel for region (%d..%d, %d..%d)\n", 
					 x0, x0 + d, y0, y0 + d);
		assert(0);
	}
}

__global__ void border_dwell2
(unsigned int* d_ns, int* d_offs1, int* d_offs2, int* dwells, int w, int h, complex cmin, complex cmax, int d, int depth, int subdiv) {
	// check whether all boundary pixels have the same dwell
    unsigned int use = blockIdx.x*SUBDIV_ELEMS2 + (blockIdx.z*gridDim.y+blockIdx.y)*2;

    const unsigned int x0 = d_offs1[use];
    const unsigned int y0 = d_offs1[use + 1];
    
    __shared__ unsigned int off_index;

	int tid = threadIdx.y * blockDim.x + threadIdx.x;
    //if (threadIdx.x > d || threadIdx.y > d){return;}
    //if (tid == 0){
    //    printf("x0y0 %i, %i\n", x0, y0);
    //}
	int bs = blockDim.x * blockDim.y;
	int comm_dwell = NEUT_DWELL;
	// for all boundary pixels, distributed across threads
	for(int r = tid; r < d; r += bs) {
		// for each boundary: b = 0 is east, then counter-clockwise
		for(int b = 0; b < 4; b++) {
			int x = b % 2 != 0 ? x0 + r : (b == 0 ? x0 + d - 1 : x0);
			int y = b % 2 == 0 ? y0 + r : (b == 1 ? y0 + d - 1 : y0);
			int dwell = pixel_dwell(w, h, cmin, cmax, x, y);
			comm_dwell = same_dwell(comm_dwell, dwell);
            //dwells[y * w + x] = 666;//pixel_dwell(w, h, cmin, cmax, x, y);
		}
	}  // for all boundary pixels
	// reduce across threads in the block
	__shared__ int ldwells[BSX * BSY];
	int nt = min(d, BSX * BSY);
	if(tid < nt)
		ldwells[tid] = comm_dwell;
	__syncthreads();
	for(; nt > 1; nt /= 2) {
		if(tid < nt / 2)
			ldwells[tid] = same_dwell(ldwells[tid], ldwells[tid + nt / 2]);
		__syncthreads();
	}
    comm_dwell = ldwells[0];

	__syncthreads();
    if(comm_dwell != DIFF_DWELL) {
        //return;
        
        int x = threadIdx.x;
        int y = threadIdx.y;
        for (int ry=y; ry < d; ry+=blockDim.y){
            for (int rx=x; rx < d; rx+=blockDim.x){
                if(rx < d && ry < d) {
                    int rxx = rx+x0, ryy = ry+y0;
                    //if (dwells[ryy * w + rxx] != 666)
                    dwells[ryy * w + rxx] = comm_dwell;
                }

            }
        }
    } else if(depth + 1 < MAX_DEPTH && d/SUBDIV > MIN_SIZE) {
        if (tid == 0){
            off_index = atomicAdd(d_ns, 1);
        }
        __syncthreads();
        if (tid < SUBDIV_ELEMS2){
                d_offs2[(off_index*SUBDIV_ELEMS2)+tid] = (x0
                        + ((tid>>1)&SUBDIV_ELEMSX)*(d/SUBDIV))*((tid+1)&1)
                    + (y0 + (tid>>SUBDIV_ELEMSP)*(d/SUBDIV))*(tid&1);
        }
    } else {
        //return;
        int x = threadIdx.x ;
        int y = threadIdx.y ;
        for (int ry=y; ry < d; ry+=blockDim.y){
            for (int rx=x; rx < d; rx+=blockDim.x){
                if(rx < d && ry < d) {
                    int rxx = rx+x0, ryy = ry+y0;
                    //if (dwells[ryy * w + rxx] != 666)
                    dwells[ryy * w + rxx] = pixel_dwell(w, h, cmin, cmax, rxx, ryy);
                }

            }
        }
    }
    //cucheck_dev(cudaGetLastError());

}  // border_dwell

void mandelbrot_pseudo_dynamic_parallelism(int *dwell, unsigned int* h_nextSize, unsigned int* d_nextSize, int* d_offsets1, int* d_offsets2, int w, int h, complex cmin, complex cmax, int d, int depth){
    
	dim3 b(BSX, BSY, 1), g(1, INIT_SUBDIV, INIT_SUBDIV);
    //printf("Running kernel with b(%i,%i) and g(%i, %i, %i) and d=%i\n", b.x, b.y, g.x, g.y, g.z, d);
    border_dwell2<<<g, b>>>(d_nextSize, d_offsets1, d_offsets2, dwell, h, w, cmin, cmax, d, depth, INIT_SUBDIV);
    for (int i=depth+1; i<MAX_DEPTH && d/SUBDIV>MIN_SIZE; i++){
        hipDeviceSynchronize();
        hipMemcpy(h_nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost);
        hipMemset(d_nextSize, 0, sizeof(int));
        std::swap(d_offsets1, d_offsets2);
        d = d/SUBDIV;
        dim3 g(*h_nextSize, SUBDIV, SUBDIV);
        //printf("Running kernel with b(%i,%i) and g(%i, %i, %i) and d=%i\n", b.x, b.y, g.x, g.y, g.z, d);
        border_dwell2<<<g, b>>>(d_nextSize, d_offsets1, d_offsets2, dwell, h, w, cmin, cmax, d, i, SUBDIV);
    }


}

/** gets the color, given the dwell (on host) */
#define CUT_DWELL (MAX_DWELL / 4)
void dwell_color(int *r, int *g, int *b, int dwell) {
	// black for the Mandelbrot set
    if (dwell == 666){
		*r = 255; 
        *g = *b = 0;
    } else if(dwell >= MAX_DWELL) {
		*r = *g = *b = 0;
	} else {
		// cut at zero
		if(dwell < 0)
			dwell = 0;
		if(dwell <= CUT_DWELL) {
			// from black to blue the first half
			*r = *g = 0;
			*b = 128 + dwell * 127 / (CUT_DWELL);
		} else {
			// from blue to white for the second half
			*b = 255;
			*r = *g = (dwell - CUT_DWELL) * 255 / (MAX_DWELL - CUT_DWELL);
		}
	}
}  // dwell_color

__global__ void mandelbrot_k
(int *dwells, int w, int h, complex cmin, complex cmax) {
	// complex value to start iteration (c)
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int dwell = pixel_dwell(w, h, cmin, cmax, x, y);
	dwells[y * w + x] = dwell;
}  // mandelbrot_k



__global__ void mandelbrot_block_k
(int *dwells, int w, int h, complex cmin, complex cmax, int x0, int y0, 
 int d, int depth) {
	x0 += d * blockIdx.x, y0 += d * blockIdx.y;
	int comm_dwell = border_dwell(dwells, w, h, cmin, cmax, x0, y0, d);
	if(threadIdx.x == 0 && threadIdx.y == 0) {
		if(comm_dwell != DIFF_DWELL) {
			// uniform dwell, just fill
			dim3 bs(BSX, BSY), grid(divup(d, BSX), divup(d, BSY));
			dwell_fill_k<<<grid, bs>>>(dwells, w, x0, y0, d, comm_dwell);
		} else if(depth + 1 < MAX_DEPTH && d / SUBDIV > MIN_SIZE) {
			// subdivide recursively
			dim3 bs(blockDim.x, blockDim.y), grid(SUBDIV, SUBDIV);
			mandelbrot_block_k<<<grid, bs>>>
				(dwells, w, h, cmin, cmax, x0, y0, d / SUBDIV, depth	+ 1);
		} else {
			// leaf, per-pixel kernel
			dim3 bs(BSX, BSY), grid(divup(d, BSX), divup(d, BSY));
			mandelbrot_pixel_k<<<grid, bs>>>
				(dwells, w, h, cmin, cmax, x0, y0, d);
		}
		//cucheck_dev(cudaGetLastError());
		//check_error(x0, y0, d);
	}
}  // mandelbrot_block_k

int checkArray(int* a, int* b, int w, int h){
    int cont = 0;
    for (int i=0; i<w*h; i++){
        if (a[i] != b[i]){
            //printf("%i, %i\n", i%h, i/h);
            cont++;
        }
    }
    return cont;
}
/** data size */
#ifndef H
#define H (32 * 1024)
#endif 

#ifndef W
#define W (32 * 1024)
#endif

#define IMAGE_PATH "./mandelbrot.png"
#define REPEATS 10

int main(int argc, char **argv) {
	// allocate memory
	int w = W, h = H;
	size_t dwell_sz = w * h * sizeof(int);
    double ti, tf, t1=0, t2=0, t3=0;

	int *h_dwells1;
	int *h_dwells2;
	int *h_dwells3;
	int *d_dwells;

	cucheck(hipMalloc((void**)&d_dwells, dwell_sz));

	h_dwells1 = (int*)malloc(dwell_sz);
	h_dwells2 = (int*)malloc(dwell_sz);
	h_dwells3 = (int*)malloc(dwell_sz);

    unsigned int *h_nextSize;
    int *h_offsets;
    unsigned int *d_nextSize;
    int *d_offsets1;
    int *d_offsets2;

    unsigned int max_elements = 2*(INIT_SUBDIV*INIT_SUBDIV)*pow(SUBDIV*SUBDIV,
            MAX_DEPTH-1)/3;
    printf("%u\n",max_elements);

    h_nextSize = (unsigned int*)malloc(sizeof(int));
	h_offsets
        = (int*)malloc(sizeof(int)*max_elements);
    for (int i=0; i<INIT_SUBDIV*INIT_SUBDIV*2; i+=2){
        h_offsets[i] = ((i/2)%INIT_SUBDIV)*(W/INIT_SUBDIV);
        h_offsets[i+1] = ((i/2)/INIT_SUBDIV)*(W/INIT_SUBDIV);

        //printf("Offsets Iniciales: (%i) - %i, %i\n", i/2, h_offsets[i], h_offsets[i+1]);
    }
    *h_nextSize = INIT_SUBDIV*INIT_SUBDIV;
	cucheck(hipMalloc((void**)&d_nextSize, sizeof(int)));

	cucheck(hipMalloc((void**)&d_offsets1, sizeof(int)*max_elements));
	cucheck(hipMalloc((void**)&d_offsets2, sizeof(int)*max_elements));

    cucheck(hipMemcpy(d_offsets1, h_offsets, sizeof(int)*max_elements, hipMemcpyHostToDevice))
    cucheck(hipMemset(d_nextSize, 0, sizeof(int)));
	// compute the dwells, copy them back


    dim3 bs(BSX, BSY), grid(divup(w, bs.x), divup(h, bs.y));

    // COMMON
    for (int i=0; i< REPEATS; i++){
        ti = omp_get_wtime();
        mandelbrot_k<<<grid, bs>>>
            (d_dwells, w, h, complex(-1.5, -1), complex(0.5, 1));
        hipDeviceSynchronize();
        tf = omp_get_wtime();
        t1 += tf - ti;
    }
    t1 /= REPEATS;

	cucheck(hipMemcpy(h_dwells1, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
    hipMemset(d_dwells, 0, dwell_sz);
    
    // 1 KERNEL
    for (int i=0; i< REPEATS; i++){
        
        for (int i=0; i<INIT_SUBDIV*INIT_SUBDIV*2; i+=2){
            h_offsets[i] = ((i/2)%INIT_SUBDIV)*(W/INIT_SUBDIV);
            h_offsets[i+1] = ((i/2)/INIT_SUBDIV)*(W/INIT_SUBDIV);

            //printf("Offsets Iniciales: (%i) - %i, %i\n", i/2, h_offsets[i], h_offsets[i+1]);
        }
        *h_nextSize = 1;
        cucheck(hipMemcpy(d_offsets1, h_offsets, sizeof(int)*max_elements, hipMemcpyHostToDevice))
        cucheck(hipMemset(d_nextSize, 0, sizeof(int)));


        ti = omp_get_wtime();
        mandelbrot_pseudo_dynamic_parallelism(d_dwells, h_nextSize, d_nextSize, d_offsets1, d_offsets2, w, h, complex(-1.5, -1), complex(0.5, 1), W / INIT_SUBDIV, 1);
        (hipDeviceSynchronize());
        tf = omp_get_wtime();
        t2 += tf - ti;
    }
    t2 /= REPEATS;

	cucheck(hipMemcpy(h_dwells2, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
    hipMemset(d_dwells, 0, dwell_sz);
	
    bs = dim3(BSX, BSY); 
    grid = dim3(INIT_SUBDIV, INIT_SUBDIV);

    // DYNAMIC PARALLELISM
    for (int i=0; i< REPEATS; i++){
        ti = omp_get_wtime();
        mandelbrot_block_k<<<grid, bs>>>(d_dwells, w, h, complex(-1.5, -1), complex(0.5, 1), 0, 0, W / INIT_SUBDIV, 1);
        (hipDeviceSynchronize());
        tf = omp_get_wtime();
        t3 += tf - ti;
    }
    t3 /= REPEATS;

	cucheck(hipMemcpy(h_dwells3, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
	
    // save the image to PNG file
	save_image("res1.png", h_dwells1, w, h);
	save_image("res2.png", h_dwells2, w, h);
	save_image("res3.png", h_dwells3, w, h);

	// print performance
    int res1 = 0;
    int res2 = 0;
    int res3 = 0;
    //printf("Check 1:\n");
    res1 = checkArray(h_dwells1, h_dwells2, W, H);
    //printf("Check 2:\n");
    res2 = checkArray(h_dwells1, h_dwells3, W, H);
    //printf("Check 3:\n");
    res3 = checkArray(h_dwells2, h_dwells3, W, H);

    if (res1 != res2){
        exit(99);
    } 

    printf("%i, %i, %i, %i, %i, %i, %i, %f, %f, %f\n", BSX, BSY, W, H, MAX_DWELL, MAX_DEPTH,
            SUBDIV, t1, t2, t3);

	// free data
	hipFree(d_dwells);
	free(h_dwells1);
	free(h_dwells2);
	free(h_dwells3);
    exit(0);
}  // main
